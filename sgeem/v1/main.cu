#include <iostream>
#include <hip/hip_runtime.h>

const int PER_THREAD_BLOCK = 256;
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
// shared memory
template<const uint M_NUM_PER_BLOCK, const uint N_NUM_PER_BLOCK, const uint K_NUM_PER_BLOCK, const uint NUM_PER_THREAD>
__global__ void sgeem(float *A, float *B, float *C, int M, int N, int K) {
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bidx = blockIdx.x;
    int bidy = blockIdx.y;
    A = A + bidy * M_NUM_PER_BLOCK * K;
    B = B + bidx * N_NUM_PER_BLOCK;
    __shared__ float a_shared[M_NUM_PER_BLOCK][K_NUM_PER_BLOCK];
    __shared__ float b_shared[K_NUM_PER_BLOCK][N_NUM_PER_BLOCK];
    float temp[NUM_PER_THREAD] = {{0.f}};
    // 取数据
    for (int i = 0; i < K; i += K_NUM_PER_BLOCK) {
        FETCH_FLOAT4(a_shared[tidy][tidx * NUM_PER_THREAD]) = FETCH_FLOAT4(A[tidy * K + i + tidx * NUM_PER_THREAD]);
        FETCH_FLOAT4(b_shared[tidy][tidx * NUM_PER_THREAD]) = FETCH_FLOAT4(B[tidy * N + i * N + tidx * NUM_PER_THREAD]);
        __syncthreads();
        for (int j = 0; j < NUM_PER_THREAD; j++) {
            for (int kk = 0; kk < K_NUM_PER_BLOCK; kk++) {
                temp[j] += a_shared[tidy][kk] * b_shared[kk][tidx * NUM_PER_THREAD + j];
            }
        }
        __syncthreads();
    }
    C = C + bidy * M_NUM_PER_BLOCK * N + bidx * N_NUM_PER_BLOCK;
    for (int i = 0; i < NUM_PER_THREAD; i++) {
        C[tidy * N + tidx * NUM_PER_THREAD + i] = temp[i];
    }
}


bool check_equal(float* res, float* c, const int n, const int m) {
    bool flag = true;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (abs(res[i * n + j] - c[i * n + j]) > 0.005) {
                printf("res[%d][%d] = %f, c[%d][%d] = %f\n", i, j, res[i * n + j], i, j, c[i * n + j]);
                flag = false;
            }
        }
    }
    return flag;
}

void sgeem_cpu(float *A, float *B, float *C, int m, int n, int k) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            for (int l = 0; l < k; l++) {
                // printf("C[%d][%d] += A[%d][%d](%f) * B[%d][%d](%f) = %f\n", i, j, i, l, A[i * k + l], l, j, B[l * n + j], C[i * n + j]);
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

void matrix_print(float *A, int n, int m) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", A[i * n + j]);
        }
        printf("\n");
    }
}

int main() {
    int n = 1024;
    int m = 1024;
    int k = 1024;
    float *A = new float[m * k];
    float *B = new float[n * k];
    float *C = new float[n * m];
    float *C_cpu = new float[n * m];
    
    // 初始化输入矩阵
    for (int i = 0; i < m * k; i++) {
        A[i] = drand48() * 2 + 0.1;
        // A[i] = 1;
    }
    for (int i = 0; i < n * k; i++) {
        B[i] = drand48() * 2 + 0.1;
        // B[i] = 1;
    }
    // 初始化输出矩阵为0
    for (int i = 0; i < n * m; i++) {
        C[i] = 0.0f;
        C_cpu[i] = 0.0f;
    }
    sgeem_cpu(A, B, C_cpu, m, n, k);
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, n * k * sizeof(float));
    hipMalloc(&d_C, n * m * sizeof(float));
    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, n * m * sizeof(float), hipMemcpyHostToDevice);
    
    const int M_PER_THREAD_BLOCK = 32;
    const int N_PER_THREAD_BLOCK = 8;
    const int NUM_PER_THREAD = 4;
    const int M_NUM_PER_BLOCK = 32;
    const int N_NUM_PER_BLOCK = 32;
    const int K_NUM_PER_BLOCK = 32;
    dim3 grid(n / 32, m / 32);
    // dim3 grid(1, 1);
    dim3 block(N_PER_THREAD_BLOCK, M_PER_THREAD_BLOCK);
    // matrix_print(A, m, k);
    // printf("\n---------\n");
    // matrix_print(B, k, n);
    // printf("\n---------\n");
    sgeem<M_NUM_PER_BLOCK, N_NUM_PER_BLOCK, K_NUM_PER_BLOCK, NUM_PER_THREAD><<<grid, block>>>(d_A, d_B, d_C, m, n, k);
    // printf("\n---------\n");
    hipDeviceSynchronize();
    // printf("\n---------\n");
    hipMemcpy(C, d_C, n * m * sizeof(float), hipMemcpyDeviceToHost);
    // printf("\n---------\n");
    // matrix_print(C, m, k);
    // printf("\n---------\n");
    // matrix_print(C_cpu, m, k);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    if (check_equal(C_cpu, C, n, m)) {
        printf("C is equal to C\n");
    } else {
        printf("C is not equal to C\n");
    }
}
