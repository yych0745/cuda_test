#include <iostream>
#include <hip/hip_runtime.h>

const int PER_THREAD_BLOCK = 256;

__device__ void warm_add(__volatile__ float *s_input, int tid) {
    s_input[tid] += s_input[tid + 32];
    s_input[tid] += s_input[tid + 16];
    s_input[tid] += s_input[tid + 8];
    s_input[tid] += s_input[tid + 4];
    s_input[tid] += s_input[tid + 2];
    s_input[tid] += s_input[tid + 1];
}

// shared memory
template<const int NUM_PER_BLOCK>
__global__ void reduce(float *input, float *output, int n) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ float s_input[PER_THREAD_BLOCK];
    input = input + bid * NUM_PER_BLOCK;
    s_input[tid] = 0;
    for (int i = 0; i < NUM_PER_BLOCK / PER_THREAD_BLOCK; i++) {
        s_input[tid] += input[tid + i * PER_THREAD_BLOCK];
    }
    __syncthreads();
    for (int i = PER_THREAD_BLOCK / 2; i > 32; i /= 2) {
        if (tid < i) {
            s_input[tid] += s_input[tid + i];
        } 
        __syncthreads();
    }
    if (tid < 32) {
        warm_add(s_input, tid);
    }
    if (tid == 0) {
        output[bid] = s_input[0];
    }
}


bool check_equal(float a, float b) {
    return abs(a - b) < 0.005;
}


int main() {
    const int N=32*1024*1024;
    float *input = (float *)malloc(N * sizeof(float));

    // int block_num = N / PER_THREAD_BLOCK / 2;
    const int block_num = 1024 * 32;
    float *cpu_output = (float *)malloc(block_num * sizeof(float));
    float *output = (float *)malloc(block_num * sizeof(float));

    for (int i = 0; i < N; i++) {
        input[i] = drand48() * 2 + 0.1;
    }

    const int NUM_PER_BLOCK = N / block_num;
    for (int i = 0; i < block_num; i++) {
            cpu_output[i] = 0;
            for (int j = 0; j < NUM_PER_BLOCK; j++) {
                cpu_output[i] += input[i * NUM_PER_BLOCK + j];
            }
        }

    float *d_input, *d_output;
    hipMalloc((void **)&d_input, N * sizeof(float));
    hipMalloc((void **)&d_output, block_num * sizeof(float));

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, block_num * sizeof(float), hipMemcpyHostToDevice);
    dim3 grid(block_num);
    dim3 block(PER_THREAD_BLOCK);
    std::cout << "NUM_PER_BLOCK " << NUM_PER_BLOCK << std::endl;
    reduce<NUM_PER_BLOCK><<<grid, block>>>(d_input, d_output, N);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(error) << std::endl;
    }
    hipDeviceSynchronize(); // 确保printf输出可见
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    bool is_equal = true;
    for (int i = 0; i < block_num; i++) {
        if (!check_equal(cpu_output[i], output[i])) {
            std::cout << "Error at block " << i << " " << cpu_output[i] << " " << output[i] << std::endl;
            is_equal = false;
        }
    }
    if (is_equal) {
        std::cout << "Test passed" << std::endl;
    } else {
        std::cout << "Test failed" << std::endl;
    }
    hipFree(d_input);
    hipFree(d_output);
    free(input);
    free(output);
    free(cpu_output);
    return 0;
}
